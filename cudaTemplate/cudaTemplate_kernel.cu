#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// CUDA Kernel
////////////////////////////////////////////////////////////////////////////////

__constant__ int const_d[CT_MEM_SIZE];

__global__ void foo(int *gid_d)
{
    
    //----------------------------PRIMERA PARTE----------------
    
    extern __shared__ int shared_mem[];

	// size of the block
    int blockSize = blockDim.x * blockDim.y * blockDim.z;

    // Para el calculo suponemos que la dimension z es como añadir bloques adicionales detras del original.
    int threads_in_2d_block = blockDim.x * blockDim.y;
    // global thread ID in thread block
    int tidb = (threadIdx.x) + (blockDim.x * threadIdx.y) + (threads_in_2d_block * threadIdx.z);

    int threads_in_2d_grid = blockSize * gridDim.x * gridDim.y;
    // global thread ID in grid
    int tidg = (blockIdx.x * blockSize + tidb + gridDim.x * blockIdx.y * blockSize + threads_in_2d_grid * blockIdx.z);
    
    //----------------------------SEGUNDA PARTE----------------

    shared_mem[tidb] = gid_d[tidg];
    
    __syncthreads();

	/* shared memory */
    shared_mem[tidb] = (tidg + const_d[tidg % CT_MEM_SIZE]);

    __syncthreads();

    gid_d[tidg] = shared_mem[tidb];
}
